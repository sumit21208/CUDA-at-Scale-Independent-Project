#include "utils.cuh"
#include <iostream>

int main(int argc, char** argv) {
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " <image|signal> <path>" << std::endl;
        return EXIT_FAILURE;
    }

    std::string type = argv[1];
    const char* path = argv[2];

    if (type == "image") {
        processImage(path);
    } else if (type == "signal") {
        processSignal(path);
    } else {
        std::cerr << "Invalid type. Use 'image' or 'signal'." << std::endl;
        return EXIT_FAILURE;
    }

    return EXIT_SUCCESS;
}
